#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>

#define NUM_THREADS 256

int* node_list;
int bin_size;
int bin_count;
int* node_size_list;

// Put any static global variables here that you will use throughout the simulation.
int blks;
int blks_bin;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__device__ void interact(particle_t* parts, int* node_list, int* node_size_list, int row, int col, int particle, int bin_size) {
    if (row >= 0 && row < bin_size && col >= 0 && col < bin_size){
        int i = row*bin_size+col;
        for (int idx = 0; idx < node_size_list[i];idx++) {
            apply_force_gpu(parts[particle], parts[node_list[i*16+idx]]);
        }
    }
}

__global__ void compute_forces_gpu(particle_t* parts, int* node_list, int* node_size_list, int bin_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < bin_size*bin_size){
        int i = tid / bin_size;
        int j = tid % bin_size;
        for (int idx = 0; idx < node_size_list[tid]; idx++) {
            int particle = node_list[16*tid+idx];
            parts[particle].ax = 0;
            parts[particle].ay = 0;
            interact(parts, node_list, node_size_list, i-1, j-1, particle, bin_size);
            interact(parts, node_list, node_size_list, i-1, j, particle, bin_size);
            interact(parts, node_list, node_size_list, i-1, j+1, particle, bin_size);
            interact(parts, node_list, node_size_list, i, j-1, particle, bin_size);
            interact(parts, node_list, node_size_list, i, j, particle, bin_size);
            interact(parts, node_list, node_size_list, i, j+1, particle, bin_size);
            interact(parts, node_list, node_size_list, i+1, j-1, particle, bin_size);
            interact(parts, node_list, node_size_list, i+1, j, particle, bin_size);
            interact(parts, node_list, node_size_list, i+1, j+1, particle, bin_size);
        }
    }
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    bin_size = (size/cutoff)+1;
    bin_count = bin_size*bin_size;
    hipMalloc(&node_list, 16*bin_count*sizeof(int));
    hipMalloc(&node_size_list, bin_count*sizeof(int));
    blks_bin = (bin_count + NUM_THREADS - 1) / NUM_THREADS;
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
}

__global__ void reset(int* node_size_list, int bin_count) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    node_size_list[tid] = (tid < bin_count) ? 0 : node_size_list[tid];
}

__global__ void rebin(particle_t* parts, int* node_list, int* node_size_list, int bin_size, int num_parts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_parts){
        int idx = (floor(parts[tid].x/cutoff) * bin_size)+floor(parts[tid].y/cutoff);
        node_list[(16*idx)+atomicAdd(&node_size_list[idx], 1)] = tid;
    }
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    reset<<<blks_bin, NUM_THREADS>>>(node_size_list, bin_count);
    rebin<<<blks, NUM_THREADS>>>(parts, node_list, node_size_list, bin_size, num_parts);
    compute_forces_gpu<<<blks_bin, NUM_THREADS>>>(parts, node_list, node_size_list, bin_size);
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}